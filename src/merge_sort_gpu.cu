#include "hip/hip_runtime.h"
#include "merge_sort_gpu.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

namespace GPU::IterativeMergeSort
{
    __device__ void merge(int* data, int* tmp, unsigned long left, unsigned long mid, unsigned long right) {
        unsigned long i, j, k;

        i = left; j = mid + 1; k = left; 

        while (i <= mid && j <= right) {
            if (data[i] <= data[j]) {
                tmp[k++] = data[i++];
            } else {
                tmp[k++] = data[j++];
            }
        }

        while (i <= mid) {
            tmp[k++] = data[i++];
        }
        
        while (j <= right) {
            tmp[k++] = data[j++];
        }

        for (i = left; i <= right; i++) {
            data[i] = tmp[i];
        }
    }

    __global__ void kernel(int* data, int* tmp, unsigned long size)
    {
        unsigned long left, mid, right;
        unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned long current_size = 1; current_size < size; size *= 2) {
            left = idx * 2 * current_size;
            mid = min(left + current_size - 1, size - 1);
            right = min(left + 2 * current_size - 1, size - 1);
        
            if (left < size) {
                merge(data, tmp, left, mid, right);
            }
        __syncthreads(); 
        }
    }

    void sort(std::vector<int>& data)
    {
        unsigned long size = data.size();
        int* dData;
        int* dTmp;

        hipMalloc((void**)&dData, size * sizeof(int));
        hipMalloc((void**)&dTmp, size * sizeof(int));

        hipMemcpy(dData, data.data(), size * sizeof(int), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
            float elapsedTime;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        kernel<<<128,1024>>>(dData, dTmp, size);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime, start, stop);

        std::cout << "Kernel execution time: " << elapsedTime << "ms\n";

        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipMemcpy(data.data(), dData, size * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(dData);
        hipFree(dTmp);
    }
}